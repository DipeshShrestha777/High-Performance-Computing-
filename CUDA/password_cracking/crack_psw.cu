#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *attempt) {
  char password1[] = "DV78";
  char password2[] = "ER87";
  char password3[] = "GS58";
  char password4[] = "TA88";

  char *d = attempt;
  char *i = attempt;
  char *p = attempt;
  char *s = attempt;
  char *psw1 = password1;
  char *psw2 = password2;
  char *psw3 = password3;
  char *psw4 = password4;

  while(*d == *psw1) {
   if(*d == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    d++;
    psw1++;
  }
    
  while(*i == *psw2) {
   if(*i == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    i++;
    psw2++;
  }

  while(*p == *psw3) {
   if(*p == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    p++;
    psw3++;
  }

  while(*s == *psw4) {
   if(*s == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    s++;
    psw4++;
  }
  return 0;

}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char e,h;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(e='0'; e<='9'; e++){
      for(h='0'; h<='9'; h++){
            password[2] = e;
            password[3] = h;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}






